#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "saxpy.h"

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
    return (working_set_size + threadsPerBlock - 1)/threadsPerBlock;
  // TODO: implement and use this interface if necessary  
}

void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
  // TODO: implement and use this interface if necessary  
}

void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
  // TODO: implement and use this interface if necessary  
}

void saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {

    const int threadsPerBlock = 512; // change this if necessary

    float *device_x;
    float *device_y;
    float *device_result;

    // Allocate device memory buffers on the GPU
    hipMalloc(&device_x, total_elems * sizeof(float));
    hipMalloc(&device_y, total_elems * sizeof(float));
    hipMalloc(&device_result, total_elems * sizeof(float));

    // Create CUDA streams
    hipStream_t* streams = new hipStream_t[partitions];
    for (int i = 0; i < partitions; i++) {
        hipStreamCreate(&streams[i]);
    }

    double startCopyH2Dtime, endCopyH2Dtime, startCopyD2Htime, endCopyD2Htime, startGPUTime, endGPUTime, timeKernel;
    // Start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();
    startCopyH2Dtime = CycleTimer::currentSeconds();  
    startGPUTime = CycleTimer::currentSeconds();

    for (int i = 0; i < partitions; i++) {
        // Compute the size of the partition
        int partition_size = total_elems / partitions;
        int partition_offset = i * partition_size;

        // Copy input arrays to the GPU using hipMemcpyAsync with the appropriate stream
        //startCopyH2Dtime = CycleTimer::currentSeconds(); 
        hipMemcpyAsync(&device_x[partition_offset], &xarray[partition_offset], partition_size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&device_y[partition_offset], &yarray[partition_offset], partition_size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        //endCopyH2Dtime = CycleTimer::currentSeconds(); 
        //timeCopyH2DAvg += (endCopyH2Dtime - startCopyH2Dtime);
        // Compute number of blocks and threads per block
        int num_blocks = (partition_size + threadsPerBlock - 1) / threadsPerBlock;
        // Run saxpy_kernel on the GPU with the appropriate stream
        saxpy_kernel<<<num_blocks, threadsPerBlock, 0, streams[i]>>>(partition_size, alpha, &device_x[partition_offset], &device_y[partition_offset], &device_result[partition_offset]);

 
        // Copy result from GPU using hipMemcpyAsync with the appropriate stream
        //startCopyD2Htime = CycleTimer::currentSeconds();
        hipMemcpyAsync(&resultarray[partition_offset], &device_result[partition_offset], partition_size * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        //endCopyD2Htime = CycleTimer::currentSeconds();
        //timeCopyD2HAvg += (endCopyD2Htime - startCopyD2Htime);
    }
    endCopyH2Dtime = CycleTimer::currentSeconds(); 
    timeCopyH2DAvg += (endCopyH2Dtime - startCopyH2Dtime);
    // Synchronize with all streams to ensure all operations have completed
    for (int i = 0; i < partitions; i++) {
        hipStreamSynchronize(streams[i]);
    }
    endGPUTime = CycleTimer::currentSeconds();
    timeKernel += (endGPUTime - startGPUTime);
    // End timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    totalTimeAvg   += overallDuration;
    timeKernelAvg += timeKernel;
    // Free memory buffers on the GPU and destroy streams
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
    for (int i = 0; i < partitions; i++) {
        hipStreamDestroy(streams[i]);
    }
    delete[] streams;
}


void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
